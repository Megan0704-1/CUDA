
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 1024

/**
 * @brief vector add function for cpu
 */
__global__ void Vecadd(int *a, int *b, int *c, int n){
    int id = threadIdx.x;
    if(id < n){
        c[id] = a[id]+b[id];
    }
}

int main(){
    int *a, *b, *c;

    hipMallocManaged(&a, SIZE*sizeof(int));
    hipMallocManaged(&b, SIZE*sizeof(int));
    hipMallocManaged(&c, SIZE*sizeof(int));

    for(int i=0; i<SIZE; i++){
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }
    
    Vecadd<<<1, SIZE>>>(a, b, c, SIZE);
    hipDeviceSynchronize();

    for(int i=0; i<SIZE; i++){
        printf("c[%d] = %d\n", i, c[i]);
    }

    hipFree(&a);
    hipFree(&b);
    hipFree(&c);
    
    return 0;
}
