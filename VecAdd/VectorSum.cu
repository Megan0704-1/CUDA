#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <iostream>

/**
 * @brief CUDA kernel for vector addition
 */
__global__ void vectorAdd(int* a, int* b, int* c, int n){
    // calculate global thread ID (tid)
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Vector boundary guard
    // what is a single thread do?
    if(tid < n){
        c[tid] = a[tid] + b[tid];
    }
}

/**
 * @brief init vector of size n to int btw 0~99
 */
void matrix_init(int* a, int n){
    for(int i=0; i<n; i++){
        a[i] = rand() % 100;
    }
}

/**
 * @brief check vector add result
 */
void error_check(int* a, int* b, int *c, int n){
    for(int i=0; i<n; i++){
        assert(c[i] = a[i]+b[i]);
    }
}


int main(){
    // vector size of 2^16 elements
    int n = 1024;

    // Host vector pointers
    int *a_hostptr, *b_hostptr, *c_hostptr;

    // Device vector pointers
    int *a_deviceptr, *b_deviceptr, *c_deviceptr;

    // size of all vec
    size_t bytes = sizeof(int) * n;

    // Allocate host memory
    a_hostptr = (int*)malloc(bytes);
    b_hostptr = (int*)malloc(bytes);
    c_hostptr = (int*)malloc(bytes);

    // Allocate device memory
    hipMalloc(&a_deviceptr, bytes);
    hipMalloc(&b_deviceptr, bytes);
    hipMalloc(&c_deviceptr, bytes);
   
    // Initialize vectors a and b with random values between 0 and 99
    matrix_init(a_hostptr, n);
    matrix_init(b_hostptr, n);

    // Copy data from host to device memory
    hipMemcpy(a_deviceptr, a_hostptr, bytes, hipMemcpyHostToDevice);
    hipMemcpy(b_deviceptr, b_hostptr, bytes, hipMemcpyHostToDevice);

    // Threadblock size : should be a multiply of 32 -> wraps is size of 32.
    int NUM_THREADS = 256;

    // Grid Size
    // launching NUMBLOCKS of Threadblocks, each of size NUM_THREADS
    int NUM_BLOCKS = (int)ceil(n/NUM_THREADS);

    // lauch kernel code on default stream without sh mem
    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(a_deviceptr, b_deviceptr, c_deviceptr, n);   
    // Copy sum vector from device to cpu
    hipMemcpy(c_hostptr, c_deviceptr, bytes, hipMemcpyDeviceToHost);

    // Check result for errors
    error_check(a_hostptr, b_hostptr, c_hostptr, n);
    
    printf("Completed\n");

    return 0;

}
