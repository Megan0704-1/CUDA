
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100000

__global__ void add(int *a, int* b, int* c){
    int cpuid=0;
    while(cpuid < N){
        c[cpuid] = a[cpuid] + b[cpuid];
        cpuid ++;
    }
}

int main(void){
    int a[N], b[N], c[N];

    // fill the arr
    for(int i=0; i<N; i++){
        a[i] = -i;
        b[i] = i*i;
    }

    add<<<1,1>>>(a, b, c);

    // diaply the result
    for(int i=N-10; i<N; i++){
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}
