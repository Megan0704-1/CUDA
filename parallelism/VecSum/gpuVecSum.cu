
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10

__global__ void add(int* a, int* b, int* c){
    int threadId = blockIdx.x;
    if(threadId < N){
        c[threadId] = a[threadId] + b[threadId];
    }
    printf("thread Id: %d, a[id]: %d\n", threadId, a[threadId]);
    return;
}

int main(void){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // allocate the memory on GPU
    hipMalloc((void**)&dev_a, N*sizeof(int));
    hipMalloc((void**)&dev_b, N*sizeof(int));
    hipMalloc((void**)&dev_c, N*sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for(int i=0; i<N; i++){
        a[i] = -i;
        b[i] = i*i;
    }

    // copy the arrays to the GPU
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // copy the array c back from the GPU to CPU
    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    // display the resule
    for(int i=0; i<N; i++){
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // free the memory allocated on GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
