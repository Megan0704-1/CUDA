#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define DIM 1000

struct hipComplex {
    float r;
    float i;

    // constructor
    hipComplex( float a, float b ): r(a), i(b) {}

    // method
    float magnitude2( void ){ return r*r + i*i; }

    hipComplex operator* (const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }

    hipComplex operator+ (const hipComplex &a){
        return hipComplex(r+a.r, i+a.i);
    }
};

int julia(int x, int y){
    const float scale = 1.5;
    // shift the complex coordinate to the image center. (0~DIM-1) -> (-DIM/2~DIM/2) with scalar value.
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex C(-1.9, 0.156);
    hipComplex a(jx, jy);

    for(int i=0; i<1000; i++){
        a = a*a + C;
        if(a.magnitude2() > 1000){
            return 0;
        }
    }
    return 1;
}

void kernel( unsigned char *ptr ){
    for(int y=0; y<DIM; y++){
        for(int x=0; x<DIM; x++){
            int offset = x + y*DIM;

            int juliaVal = julia(x, y);
            ptr[offset*4 + 0] = 255 * juliaVal;
            ptr[offset*4 + 1] = 0;
            ptr[offset*4 + 2] = 0;
            ptr[offset*4 + 3] = 255;
        }
    }
}

int main( void ){
    CPUBitmap bitmap( DIM, DIM );
    unsigned char *ptr = bitmap.get_ptr();

    kernel(ptr);

    bitmap.display_and_exit();
}
