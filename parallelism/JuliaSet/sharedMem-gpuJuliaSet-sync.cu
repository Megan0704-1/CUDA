#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define DIM 1024
#define PI 3.1415926535897932f

using uc = unsigned char;

__global__ void kernel (uc *ptr){

    __shared__ float shareMem[16][16];

    // map from threadIdx/blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // true position of the thread
    int offset = x + y * blockDim.x * gridDim.x;

    const float period = 128.0f;

    shareMem[threadIdx.x][threadIdx.y] = 255 * (sinf(x*2.0f*PI / period) + 1.0f) * 
                                               (sinf(x*2.0f*PI / period) + 1.0f) / 4.0f;

    __syncthreads();

    ptr[offset*4 + 0] = 0;
    ptr[offset*4 + 1] = shareMem[15 - threadIdx.x][15 - threadIdx.y];
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

int main(void){
    CPUBitmap bitmap( DIM, DIM );

    uc *dev_bitmap;

    hipMalloc( (void**)&dev_bitmap, bitmap.image_size() );

    dim3 grids(DIM/16, DIM/16);
    dim3 threads(16, 16);
    kernel<<<grids, threads>>>( dev_bitmap );

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

    bitmap.display_and_exit();

    hipFree(dev_bitmap);
}
