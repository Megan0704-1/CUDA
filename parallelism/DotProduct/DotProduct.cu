
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 100000000

__global__ void dotProduct(int* a, int* b, int* c){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N){
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(){
    int* a;
    int* b;
    int* c;

    int* device_a;
    int* device_b;
    int* device_c;

    a = (int*)malloc(N * sizeof(int));
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));

    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = -i;
    }

    hipMalloc((void**)&device_a, sizeof(int)*N);
    hipMalloc((void**)&device_b, sizeof(int)*N);
    hipMalloc((void**)&device_c, sizeof(int)*N);

    hipMemcpy(device_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

    dotProduct<<<128, 128>>>(device_a, device_b, device_c);

    hipMemcpy(c, device_c, sizeof(int)*N, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        assert(c[i]==a[i] * b[i]);
    }
    printf("PASSED\n");

    free(a);
    free(b);
    free(c);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    return 0;

}
