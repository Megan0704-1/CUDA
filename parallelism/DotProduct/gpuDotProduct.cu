#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../../common/book.h"

#define imin(a, b) (a<b?a:b)
#define sum_square(x) (x*(x+1)*(2*x+1)/6)

const int N = 33*1024;
const int threadPerBlock = 256;
const int blockPerGrid = imin(32, (N + threadPerBlock-1) / threadPerBlock);

__global__ void dot(float* a, float* b, float* c){
    // qualifier `__shared__` is init once per block.
    // that is, only one thread would possess this instruction for allocating memory
    __shared__ float cache[threadPerBlock];

    int cacheIdx = threadIdx.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    float cacheVal = 0;
    while(tid < N){
        cacheVal += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIdx] = cacheVal;

    // synchronizing threads in a block.
    __syncthreads();

    int i = blockDim.x/2;
    while(i!=0){
        if(cacheIdx < i){
            cache[cacheIdx] += cache[cacheIdx + i];
        }
        __syncthreads();
        i /= 2;
    }

    if(cacheIdx==0){
        c[blockIdx.x] = cache[0];
    }
}

int main(void){
    float* a, *b, *partial_c;
    float* dev_a, *dev_b, *dev_partial_c;
    float cum;

    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    partial_c = (float*)malloc(N*sizeof(float));

    hipMalloc( (void**)&dev_a, N*sizeof(float) );
    hipMalloc( (void**)&dev_b, N*sizeof(float) );
    hipMalloc( (void**)&dev_partial_c, N*sizeof(float) );
    
    for(int i=0; i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    dot<<<blockPerGrid, threadPerBlock>>>(dev_a, dev_b, dev_partial_c);

    hipMemcpy(partial_c,dev_partial_c, blockPerGrid*sizeof(float), hipMemcpyDeviceToHost);

    cum=0;
    for(int i=0; i<blockPerGrid; i++){
        cum += partial_c[i];
    }

    printf("Does GPU value %.6g = %.6g?\n", cum, 2*sum_square( (float)(N-1) ));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    free(a);
    free(b);
    free(partial_c);

    return 0;
}
