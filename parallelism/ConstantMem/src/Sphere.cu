#include "hip/hip_runtime.h"
#include "../include/Sphere.h"

__device__ float Sphere::hit(float px, float py, float* n){
    float dx = px - x;
    float dy = py - y;
    if( dx*dx + dy*dy < radius*radius ){
        float dz = sqrtf( radius*radius - dx*dx - dy*dy );
        *n = dz / sqrtf( radius*radius );
        return dz + z;
    }

    return -INF;
}
