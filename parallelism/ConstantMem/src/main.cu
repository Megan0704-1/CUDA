#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../include/Sphere.h"
#include "../../../common/book.h"
#include "../../../common/cpu_bitmap.h"


using uc = unsigned char;

__global__ void kernel( uc* ptr, Sphere* s){
    // map to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // shift (x, y) => image coordinate, by DIM/2, so that the z-axis runs through the center of the image
    float px = ( x - DIM/2 );
    float py = ( y - DIM/2 );

    float r = 0;
    float g = 0;
    float b = 0;
    float maxZ = -INF;
    for(int i=0; i<numSPHERES; i++){
        float n;
        float t = s[i].hit(px, py, &n);
        if(t > maxZ){
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
        }
    }

    ptr[offset*4 + 0] = (int)(r*255);
    ptr[offset*4 + 1] = (int)(g*255);
    ptr[offset*4 + 2] = (int)(b*255);
    ptr[offset*4 + 3] = 255;
}

int main(void){
    // capture the start time
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    CPUBitmap bitmap( DIM, DIM );
    uc *dev_bitmap;
    Sphere* s;

    // allocate space for output (dev_map), and input (s) on GPU
    hipMalloc( (void**)&dev_bitmap, bitmap.image_size() );
    hipMalloc( (void**)&s, sizeof(Sphere)*numSPHERES );

    // generate 20 spheres
    Sphere* host_s = (Sphere*)malloc( sizeof(Sphere)*numSPHERES );
    for(int i=0; i<numSPHERES; i++){
        host_s[i].r = rnd( 1.0f );
        host_s[i].g = rnd( 1.0f );
        host_s[i].b = rnd( 1.0f );
        host_s[i].x = rnd( 1000.0f ) - 500;
        host_s[i].y = rnd( 1000.0f ) - 500;
        host_s[i].z = rnd( 1000.0f ) - 500;
        host_s[i].radius = rnd( 100.0f ) + 20;
    }

    // move input to GPU
    hipMemcpy(s, host_s, sizeof(Sphere)*numSPHERES, hipMemcpyHostToDevice);
    free(host_s);

    // generate a bitmap from sphere data
    dim3 grids( DIM/16, DIM/16 );
    dim3 threads( 16, 16 );
    kernel<<<grids, threads>>>(dev_bitmap, s);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size() , hipMemcpyDeviceToHost);
    
    bitmap.display_and_exit();

    // free out memory
    hipFree(dev_bitmap);
    hipFree(s);
}
