#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "../include/Sphere.h"
#include "../../../common/book.h"
#include "../../../common/cpu_bitmap.h"

using uc=unsigned char;

// declare constant memory allocation
__constant__ Sphere s[numSPHERES];

__global__ void kernel(uc* ptr){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
    float px = x - (DIM/2);
    float py = y - (DIM/2);

    float r=0, g=0, b=0;
    float maxZ = -INF;

    for(int i=0; i<numSPHERES; i++)
    {
        float n; // n is the scale of color
        float dist = s[i].hit(px, py, &n);
        if(dist > maxZ)
        {
            float scale = n;
            r = s[i].r * scale;
            g = s[i].g * scale;
            b = s[i].b * scale;
        }
        
    }

    ptr[offset*4 + 0] = (int)(255 * r);
    ptr[offset*4 + 1] = (int)(255 * g);
    ptr[offset*4 + 2] = (int)(255 * b);
    ptr[offset*4 + 3] = (int)(255);
}

int main(void)
{
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

    CPUBitmap bitmap( DIM, DIM );
    uc *dev_bitmap;

    hipMalloc( (void**)&dev_bitmap, bitmap.image_size() );
    

    Sphere* host_s = (Sphere*)malloc( sizeof(Sphere)*numSPHERES );
    for( int i=0; i<numSPHERES; i++ )
    {
        host_s[i].r = rnd( 1.0f );
        host_s[i].g = rnd( 1.0f );
        host_s[i].b = rnd( 1.0f );
        host_s[i].x = rnd( 1000.0f ) - 500;
        host_s[i].y = rnd( 1000.0f ) - 500;
        host_s[i].z = rnd( 1000.0f ) - 500;
        host_s[i].radius = rnd( 100.0f ) + 20;
    }

    // different memcpy api
    hipMemcpyToSymbol(HIP_SYMBOL( s), host_s, sizeof(Sphere)*numSPHERES );

    free(host_s);
    
    // num of blocks in grids
    dim3 blocks( DIM/16, DIM/16 );
    // num of threads per block
    dim3 threads( 16, 16 );

    kernel<<<blocks, threads>>>( dev_bitmap);

    hipMemcpy(bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost);

    bitmap.display_and_exit();

    hipFree( dev_bitmap );
}
