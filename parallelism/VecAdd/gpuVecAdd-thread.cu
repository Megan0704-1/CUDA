
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#define N 1000
#define MAX_ERROR 1e-6

__global__ void add(float* a, float* b, float* c, int n){
    int idx = threadIdx.x;
    int stride = blockDim.x;

    printf("thread id: %d, block id: %d\n", idx, stride);
    for(int i=idx; i<n; i+=stride){
        c[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    c = (float*)malloc(sizeof(float)*N);

    for(int i=0; i<N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMalloc((void**)&dev_a, sizeof(float)*N);
    hipMalloc((void**)&dev_b, sizeof(float)*N);
    hipMalloc((void**)&dev_c, sizeof(float)*N);

    hipMemcpy(dev_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    add<<<1, 256>>>(dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, sizeof(float)*N, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        assert(fabs(c[i] - a[i] - b[i]) < MAX_ERROR);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
