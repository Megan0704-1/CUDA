
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>

#define N 100000000
#define MAX_ERROR 1e-6

__global__ void add(float* a, float* b, float* c, int n){
    int idx = threadIdx.x;
    int stride = blockDim.x;
    int blockID = blockIdx.x;

    int tid = idx + stride * blockID;

    while(tid < n){
        c[tid] = a[tid] + b[tid];
        tid += stride * gridDim.x;
    }
}

int main(){
    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;

    a = (float*)malloc(sizeof(float)*N);
    b = (float*)malloc(sizeof(float)*N);
    c = (float*)malloc(sizeof(float)*N);

    for(int i=0; i<N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    hipMalloc((void**)&dev_a, sizeof(float)*N);
    hipMalloc((void**)&dev_b, sizeof(float)*N);
    hipMalloc((void**)&dev_c, sizeof(float)*N);

    hipMemcpy(dev_a, a, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(float)*N, hipMemcpyHostToDevice);

    add<<<N/256+1, 256>>>(dev_a, dev_b, dev_c, N);

    hipMemcpy(c, dev_c, sizeof(float)*N, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
        assert(fabs(c[i] - a[i] - b[i]) < MAX_ERROR);
    }

    printf("PASSED\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);

    return 0;
}
